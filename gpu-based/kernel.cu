#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstring>
#include <ctime>
#include <math.h>
#include <time.h>
#include <random>
#include <chrono>
#include <vector>
#include <string>
#include <sstream>

#include "classes.cpp"

/* CUDA Runtime */
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

__host__ void check_CUDA_error(const char* msg) {
	hipError_t err;
	hipDeviceSynchronize();
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error (%d): [%s] %s\n", err, msg, hipGetErrorString(err));
	}
}

using namespace std;

class Agent;
class Operations;
class GlobalState;

typedef std::chrono::high_resolution_clock myclock;
myclock::time_point beginning = myclock::now();
myclock::duration d = myclock::now() - beginning;
unsigned seed2 = d.count();
std::mt19937 rng(seed2);
std::default_random_engine generator;

__global__ void randomCudaGenerator(int min, int max, double* result)
{
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	hiprandState state;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

	double rand1 = hiprand_uniform_double(&state) * (max - min) + min;
	*result = rand1;
	printf("randomCudaGenerator :: %f\n", rand1);
}

 float GenerateRandomFloatBetween(float a, float b)
{
	float res;
	std::uniform_real_distribution<double> distribution(a, b);
	res = distribution(generator);

	return res;
}
 int GenerateRandomIntegerBetween(int a, int b)
{
	std::uniform_int_distribution<int> gen(a, b);
	return gen(rng);
}

/*Operations Declarations*/
//First Operation
int hasBeenInfected(Agent* agent, vector<Agent*>* AgentList)
{
	if (agent->S != 0) {
		return agent->S;
	}
	// Part First Part
	float prob = GenerateRandomFloatBetween(0, 1);
	bool fp = prob <= agent->Pcon ? 1 : 0;
	bool a = 0;

	// Part: Calculate Alpha
	bool sp = 0;
	bool tp = 0;
	int sum = 0;

	for (Agent* agent_test : *AgentList)
	{
		if (agent->x == agent_test->x && agent->y == agent_test->y)
		{
			continue;
		}

		float distance = (float)sqrt(pow(agent_test->x - agent->x, 2) + pow(agent_test->y - agent->y, 2));
		sp = distance <= GlobalState::R;
		// Part: Calculate Beta
		tp = agent_test->S > 0 ? 1 : 0;
		sum += sp * tp;
	}
	a = sum >= 1 ? 1 : 0;


	return (fp * a) ? 1 : agent->S;
}

//Second Operation
int isShortMovement(Agent* agent)
{
	float prob = GenerateRandomFloatBetween(0, 1);
	return (prob <= agent->Psmo) ? 1 : 0;
}

int XMovement(Agent* agent)
{
	int s = isShortMovement(agent);
	int p = GlobalState::mapSize[1];
	float prob = GenerateRandomFloatBetween(0, 1);
	// printf("prob X 1: %f\n", prob);
	float longDistance = p * prob * (1 - s);
	prob = GenerateRandomFloatBetween(0, 1);
	// printf("prob X 2: %f\n", prob);
	float shortDistance = (agent->x + (2 * prob - 1) * GlobalState::lmax) * s;
	return shortDistance + longDistance;
}

int YMovement(Agent* agent)
{
	int s = isShortMovement(agent);
	int q = GlobalState::mapSize[0];
	float prob = GenerateRandomFloatBetween(0, 1);
	// printf("prob Y 1: %f\n", prob);
	float longDistance = q * prob * (1 - s);
	prob = GenerateRandomFloatBetween(0, 1);
	// printf("prob Y 2: %f\n", prob);
	float shortDistance = (agent->y + (2 * prob - 1) * GlobalState::lmax) * s;
	return shortDistance + longDistance;
}

bool willMove(Agent* agent)
{
	float prob = GenerateRandomFloatBetween(0, 1);
	return (prob <= agent->Pmov) ? 1 : 0;
}

//Third Operation
int hasBeenExternalInfected(Agent* agent)
{
	float prob = GenerateRandomFloatBetween(0, 1);
	//Part: Calculate First Part
	bool fp = prob <= agent->Pext ? 1 : 0;
	//Part: Calcula Epsilon
	bool sp = agent->S != 0 ? 0 : 1;
	//fp * sp == if fp or sp are 0 the condition is false
	bool finalB = (fp * sp) > 0;

	return (finalB) ? 1 : agent->S;
}

//Third Operation
__device__ int hasBeenExternalInfected_GPU(Agent* agent, float prob)
{
	//float prob = GenerateRandomFloatBetween(0, 1);
	//Part: Calculate First Part
	bool fp = prob <= agent->Pext ? 1 : 0;
	//Part: Calcula Epsilon
	bool sp = agent->S != 0 ? 0 : 1;
	//fp * sp == if fp or sp are 0 the condition is false
	bool finalB = (fp * sp) > 0;

	return (finalB) ? 1 : agent->S;
}

//Fourth Operation
__device__ int incubationTime(Agent* agent)
{
	int Tinc = agent->Tinc;
	if (agent->S > 0)
	{
		Tinc--;
	}
	return Tinc;
}

__device__ int hasSymptoms(Agent* agent)
{
	int S = -1;
	if (agent->Tinc > 0)
	{
		S = agent->S;
	}
	return S;
}

__device__ int recuperationTime(Agent* agent)
{
	int Trec = agent->Trec;
	if (agent->S == -1)
	{
		Trec--;
	}
	return Trec;
}

//Fifth Operation
 int isInRecuperation(Agent* agent)
{
	return (agent->S < 0) ? 1 : 0;
}
__device__ int isInRecuperation_GPU(Agent* agent)
{
	return (agent->S < 0) ? 1 : 0;
}

int isDead(Agent* agent)
{
	int o = isInRecuperation(agent);
	float prob = GenerateRandomFloatBetween(0, 1);
	int Pfat = (prob <= agent->Pfat) ? 1 : 0;
	return (Pfat * o > 0) ? -2 : agent->S;
}

__device__ int isDead_GPU(Agent* agent, float prob)
{
	int o = isInRecuperation_GPU(agent);
	//float prob = GenerateRandomFloatBetween(0, 1);
	int Pfat = (prob <= agent->Pfat) ? 1 : 0;
	return (Pfat * o > 0) ? -2 : agent->S;
}

int Clamp(int value, int low, int high)
{
	return value > high ? high : value < low ? low : value;
}

/*GlobalState Declarations*/
int GlobalState::mapSize[2] = { 500, 500 };
int GlobalState::Dmax = 60; //// DIAS
int GlobalState::Mmax = 10;
int GlobalState::lmax = 5;
int GlobalState::N = 10240; //// AGENTES 
// int GlobalState::N = 5000;
int GlobalState::infectedAgents = 0;
int GlobalState::R = 1;
/* Statistics*/
int GlobalState::curedAgents = 0;
int GlobalState::deadAgents = 0;
int GlobalState::patientZeroDay = 0;
int GlobalState::halfPopulationInfectedDay = 0;
int GlobalState::fullPopulationInfectedDay = 0;
int GlobalState::patientZeroCuredDay = 0;
int GlobalState::halfPopulationCuredDay = 0;
int GlobalState::fullPopulationCuredDay = 0;
int GlobalState::patientZeroDeadDay = 0;
int GlobalState::halfPopulationDeadDay = 0;
int GlobalState::fullPopulationDeadDay = 0;

float Agent::PconRange[2] = { 0.02, 0.03 };
float Agent::PextRange[2] = { 0.02, 0.03 };
float Agent::PfatRange[2] = { 0.007, 0.07 };
float Agent::PmovRange[2] = { 0.3, 0.5 };
float Agent::PsmoRange[2] = { 0.7, 0.9 };
int Agent::TincRange[2] = { 5, 6 };

/*Agent Constructor*/
Agent::Agent(int x, int y, int i)
{
	// Position
	Agent::id = i;
	Agent::x = x;
	Agent::y = y;
	// Assing the Infection Probability
	Agent::Pcon = GenerateRandomFloatBetween(Agent::PconRange[0], Agent::PconRange[1]);
	// Assign the External Infection Probability
	Agent::Pext = GenerateRandomFloatBetween(Agent::PextRange[0], Agent::PextRange[1]);
	//Assign the Mortality Probability
	Agent::Pfat = GenerateRandomFloatBetween(Agent::PfatRange[0], Agent::PfatRange[1]);
	// Assign the Movement Probability
	Agent::Pmov = GenerateRandomFloatBetween(Agent::PmovRange[0], Agent::PmovRange[1]);
	// Assign the Small Movement Probability
	Agent::Psmo = GenerateRandomFloatBetween(Agent::PsmoRange[0], Agent::PsmoRange[1]);
	// Assign the Incubation Time
	Agent::Tinc = GenerateRandomFloatBetween(Agent::TincRange[0], Agent::TincRange[1]);
};

__global__ void Rule345(Agent* AgentList) {

	int min_1 = 0;
	int max_1 = 1;
	int min_2 = 0;
	int max_2 = 1;
	hiprandState state;
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
	double rand1 = hiprand_uniform_double(&state) * (max_1 - min_1) + min_1;
	double rand2 = hiprand_uniform_double(&state) * (max_2 - min_2) + min_2;
	int gId = blockIdx.x * blockDim.x + threadIdx.x;

	Agent agent = AgentList[gId];

	//Aplicar Regla 3
	//agent->S = hasBeenExternalInfected(agent);
	agent.S = hasBeenExternalInfected_GPU(&agent, (float)rand1);

	//Aplicar Regla 4
	agent.Tinc = incubationTime(&agent);
	agent.S = hasSymptoms(&agent);
	agent.Trec = recuperationTime(&agent);
	if (agent.Trec <= 0) {
		agent.Pcon = -1;
		agent.Pext = -1;
		agent.Pfat = -1;
		agent.S = 0;

		if (!agent.hasBeenAccountedFor) {
			agent.hasBeenAccountedFor = true;
		}

	}
	//Aplicar Regla 5
	agent.S = isDead_GPU(&agent, (float)rand2);

	AgentList[gId] = agent;
}


/**/
int main()
{
	clock_t total_start_CPU = clock();

	/* 1. Map Generation and Setup */
	int size0 = 500;
	int size1 = 500;
	vector<int> historyInfectedPerDay;
	vector<int> historyCuredPerDay;
	vector<int> historyDeadPerDay;

	//vector<Agent*> AgentList;
	//Agent** AgentList = new Agent*[GlobalState::N];
	Agent* AgentList = new Agent[GlobalState::N];
	Agent **AgentMap = new Agent*[500 * 500];


		for (int i = 0; i < size0; i++)
		{
			for (int j = 0; j < size1; j++)
			{
				AgentMap[i * 500 + j] = NULL;
			}
		}


	/* 2. Generate Agents on Map */
	int leftAgents = GlobalState::N;
	int i = 0;
	while (leftAgents > 0)
	{
		int x = GenerateRandomIntegerBetween(0, 499);
		int y = GenerateRandomIntegerBetween(0, 499);

		if (AgentMap[y * 500 + x] != NULL)
		{
			continue;
		}

		Agent* agent = new Agent(x, y, i);
		// printf("(%d,%d)\n", agent->x, agent->y);
		AgentMap[y * 500 + x] = agent;

		AgentList[i] = *agent;
		leftAgents--;
		i++;
	}

	/* Operation Phase*/
	int currentDay = 0;
	//Mientras que el d�a actual de simulaci�n sea menor que dMax
	while (currentDay < GlobalState::Dmax)
	{
		int infectedToday = 0;
		int killedToday = 0;
		int curedToday = 0;

		int currentMovement = 0;
		//Mientras que el movimiento actual sea menor que mMax
		while (currentMovement < GlobalState::Mmax)
		{
			//Para todos los agentes
			for (int i = 0; i < GlobalState::N; i++)
			{
				Agent* agent = &AgentList[i];
				int previousState = agent->S;

				//Aplicar Regla 1
				vector<Agent*> Neighbours;
				//if (agent->y > 0 && AgentMap[agent->y - 1][agent->x] != NULL)
				if (agent->y > 0 && AgentMap[(agent->y - 1)*500+(agent->x)] != NULL)
				{
					Neighbours.push_back(AgentMap[(agent->y - 1) * 500 + (agent->x)]);
				}
				if (agent->x > 0 && AgentMap[(agent->y)*500 + (agent->x - 1)] != NULL)
				{
					Neighbours.push_back(AgentMap[(agent->y) * 500 + (agent->x - 1)]);
				}
				if (agent->x < size0 - 1 && AgentMap[(agent->y) * 500 + (agent->x + 1)] != NULL)
				{
					Neighbours.push_back(AgentMap[(agent->y) * 500 + (agent->x + 1)]);
				}
				if (agent->y < size1 - 1 && AgentMap[(agent->y + 1)*500 + (agent->x)] != NULL)
				{
					Neighbours.push_back(AgentMap[(agent->y + 1) * 500 + (agent->x)]);
				}
				agent->S = hasBeenInfected(agent, &Neighbours);


				//Aplicar Regla 2
				if (willMove(agent))
				{
					int x;
					int y;

					do
					{
						x = XMovement(agent);
						y = YMovement(agent);

						x = Clamp(x, 0, size0 - 1);
						y = Clamp(y, 0, size1 - 1);
					} while (AgentMap[y * 500 + x] != NULL);

					//Se elimina de la vieja posici�n
					AgentMap[agent->y * 500 + agent->x] = NULL;

					//Se a�ade a la nueva posici�n
					agent->x = x;
					agent->y = y;
					AgentMap[y * 500 + x] = agent;
				}

				if (agent->S != previousState) {
					switch (agent->S) {
					case 1: //Infected
					  // printf("Agent(%d, %d) has been infected on Internal!\n", agent->x, agent->y);
						if (GlobalState::infectedAgents == 0)
						{
							printf("\t\tPatient Zero has been found\n");
							GlobalState::patientZeroDay = currentDay;
						}
						GlobalState::infectedAgents++;
						infectedToday++;
						break;
					}
				}
			}

			currentMovement++;
		}

		/* KERNEL 345 */

		Agent* DEV_AgentList;
		Agent* DEV_Result_AgentList;
		DEV_Result_AgentList = (Agent*)malloc(GlobalState::N * sizeof(Agent));

		hipMalloc((void**)&DEV_AgentList, GlobalState::N * sizeof(Agent));
		check_CUDA_error("Malloc DEV_AgentList :: 345");

		hipMemcpy(DEV_AgentList, AgentList, GlobalState::N * sizeof(Agent), hipMemcpyHostToDevice);
		check_CUDA_error("Memcpy Data HOST :: DEV 345");

		dim3 grid(10);
		dim3 block(1024);

		Rule345 << <grid, block >> > (DEV_AgentList);
		check_CUDA_error("Rule345 ::");

		hipMemcpy(DEV_Result_AgentList, DEV_AgentList, GlobalState::N * sizeof(Agent), hipMemcpyDeviceToHost);
		check_CUDA_error("Memcpy Data DEV :: HOST 345");

		int infectedAgentsGPU = 0;
		int curedAgentsGPU = 0;
		int deadAgentsGPU = 0;

		int totalInfected = 0;

		for (int i = 0; i < GlobalState::N; i++) {
			Agent* gpu_agent = &DEV_Result_AgentList[i];

			//printf("[Pos:%d] Agent [ID:%d] (%d, %d) -> Status: %d\n", i, gpu_agent->id, gpu_agent->x, gpu_agent->y, gpu_agent->S);
			switch (gpu_agent->S)
			{
			case 1: 
				//Infected

				infectedAgentsGPU++;
				totalInfected++;
				break;
			case -1:
				totalInfected++;
				break;
			case -2:
				//Dead
				gpu_agent->Pcon = -1;
				gpu_agent->Pext = -1;
				gpu_agent->Pfat = -1;
				gpu_agent->S = 0;
				if (!gpu_agent->hasBeenAccountedFor) {
					deadAgentsGPU++;
					gpu_agent->hasBeenAccountedFor = true;
					gpu_agent->hasDied = true;
				}
				totalInfected++;
				break;
			case 0:
				if (gpu_agent->hasBeenAccountedFor && !gpu_agent->hasDied) {
					curedAgentsGPU++;
					totalInfected++;
				}
				break;
			}
		}
		
		/*printf("\tB:Infected: %d\n", infectedToday);
		printf("\tB:Dead: %d\n", killedToday);
		printf("\tB:Cured: %d\n\n", curedToday);

		printf("\tTotalInfectedGPU: %d\n", totalInfected);
		printf("\tTotalDeadGPU: %d\n", deadAgentsGPU);
		printf("\tTotalCuredGPU: %d\n\n", curedAgentsGPU);

		printf("\tGlobalInfected: %d\n", GlobalState::infectedAgents);
		printf("\tGlobalDead: %d\n", GlobalState::deadAgents);
		printf("\tGlobalCured: %d\n\n", GlobalState::curedAgents);

		printf("\tCalcInfectedToday: %d\n", abs(GlobalState::infectedAgents - totalInfected));
		printf("\tCalcDeadToday: %d\n", abs(GlobalState::deadAgents - deadAgentsGPU));
		printf("\tCalcCuredToday: %d\n\n", abs(GlobalState::curedAgents - curedAgentsGPU));*/

		infectedToday += totalInfected - GlobalState::infectedAgents;
		if (infectedToday < 0) {
			infectedToday = 0;
		}
		killedToday += deadAgentsGPU - GlobalState::deadAgents;
		if (killedToday < 0) {
			killedToday = 0;
		}
		curedToday += curedAgentsGPU - GlobalState::curedAgents;
		if (curedToday < 0) {
			curedToday = 0;
		}

		historyInfectedPerDay.push_back(infectedToday);
		historyDeadPerDay.push_back(killedToday);
		historyCuredPerDay.push_back(curedToday);

		// printf("\tInfected: %d\n", infectedToday);
		// printf("\tDead: %d\n", killedToday);
		// printf("\tCured: %d\n", curedToday);


		int sumInfected = 0;
		int sumKilled = 0;
		int sumCured = 0;
		for (int i = 0; i < historyInfectedPerDay.size(); i++) {
			sumInfected += historyInfectedPerDay[i];
		}
		for (int i = 0; i < historyDeadPerDay.size(); i++) {
			sumKilled += historyDeadPerDay[i];
		}
		for (int i = 0; i < historyCuredPerDay.size(); i++) {
			sumCured += historyCuredPerDay[i];
		}

		GlobalState::infectedAgents = sumInfected;
		GlobalState::deadAgents = sumKilled;
		GlobalState::curedAgents = sumCured;

		AgentList = DEV_Result_AgentList;
		currentDay++;
	}	
	clock_t total_end_CPU = clock();
	float total_elapsedTime_CPU = total_end_CPU - total_start_CPU;

	int maxInfected = GlobalState::infectedAgents;
	int sumInfected = 0;
	int sumDead = 0;
	int sumCured = 0;

	bool hasFirstInfectedBeenDetected = false;
	bool hasFirstDeadBeenDetected = false;
	bool hasFirstCuredBeenDetected = false;
	
	bool hasHalfPopulationInfectedBeenDetected = false;
	bool hasHalfPopulationDeadBeenDetected = false;
	bool hasHalfPopulationCuredBeenDetected = false;

	bool hasAllPopulationInfectedBeenDetected = false;
	bool hasAllPopulationDeadBeenDetected = false;
	bool hasAllPopulationCuredBeenDetected = false;

	for(int i = 0; i < GlobalState::Dmax; i++){
		int curDayInfected  = historyInfectedPerDay[i];
		int curDayDead  = historyDeadPerDay[i];
		int curDayCured  = historyCuredPerDay[i];

		sumInfected += curDayInfected;
		sumDead += curDayDead;
		sumCured += curDayCured;

		/* Patients Zero */    
		if(!hasFirstInfectedBeenDetected){
		if(curDayInfected > 0){
			GlobalState::patientZeroDay = i+1;
			hasFirstInfectedBeenDetected = true;
		}
		}
		if(!hasFirstDeadBeenDetected){
		if(curDayDead > 0){
			GlobalState::patientZeroDeadDay = i+1;
			hasFirstDeadBeenDetected = true;
		}
		}
		if(!hasFirstCuredBeenDetected){
		if(curDayCured > 0){
			GlobalState::patientZeroCuredDay = i+1;
			hasFirstCuredBeenDetected = true;
		}
		}

		/* Half population */
		if(!hasHalfPopulationInfectedBeenDetected){
		if(sumInfected > (maxInfected/2) ){
			GlobalState::halfPopulationInfectedDay = i+1;
			hasHalfPopulationInfectedBeenDetected = true;
		}
		}
		if(!hasHalfPopulationDeadBeenDetected){
		if(sumDead > (maxInfected/2) ){
			GlobalState::halfPopulationDeadDay = i+1;
			hasHalfPopulationDeadBeenDetected = true;
		}
		}
		if(!hasHalfPopulationCuredBeenDetected){
		if(sumCured > (maxInfected/2) ){
			GlobalState::halfPopulationCuredDay = i+1;
			hasHalfPopulationCuredBeenDetected = true;
		}
		}

		/* All population */
		if(!hasAllPopulationInfectedBeenDetected){
		if(sumInfected >= maxInfected ){
			GlobalState::fullPopulationInfectedDay = i+1;
			hasAllPopulationInfectedBeenDetected = true;
		}
		}
		if(!hasAllPopulationDeadBeenDetected){
		if(sumDead >= maxInfected ){
			GlobalState::fullPopulationDeadDay = i+1;
			hasAllPopulationDeadBeenDetected = true;
		}
		}
		if(!hasAllPopulationCuredBeenDetected){
		if(sumCured >= maxInfected ){
			GlobalState::fullPopulationCuredDay = i+1;
			hasAllPopulationCuredBeenDetected = true;
		}
		}

	}

	/* END */
	printf("Phase End\n");
	printf("\tTotal Time GPU: %f ms.\n", total_elapsedTime_CPU);
	printf("===\nTotal Agents: %d\n===\n", GlobalState::N);
	printf("Total Infected Agents: %d\n", GlobalState::infectedAgents);
	printf("Total Dead Agents: %d\n", GlobalState::deadAgents);
	printf("Total Cured Agents: %d\n===\n", GlobalState::curedAgents);
	printf("Day of First Agent Infection: %d\n", GlobalState::patientZeroDay);
	printf("Day of First Agent Dead: %d\n", GlobalState::patientZeroDeadDay);
	printf("Day of First Agent Cured: %d\n===\n", GlobalState::patientZeroCuredDay);
	printf("Day of Half Population Infection: %d\n", GlobalState::halfPopulationInfectedDay);
	printf("Day of Half Population Dead: %d\n", GlobalState::halfPopulationDeadDay);
	printf("Day of Half Population Cured: %d\n===\n", GlobalState::halfPopulationCuredDay);
	printf("Day of Full Population Infection: %d\n", GlobalState::fullPopulationInfectedDay);
	printf("Day of Full Population Dead: %d\n", GlobalState::fullPopulationDeadDay);
	printf("Day of Full Population Cured: %d\n===\n", GlobalState::fullPopulationCuredDay);

	printf("Agents infected per Day (History):");
	for (int val : historyInfectedPerDay)
	{
		printf(" %d", val);
	}
	printf("\n");
	printf("Agents killed per Day (History):");
	for (int val : historyDeadPerDay)
	{
		printf(" %d", val);
	}
	printf("\n");
	printf("Agents cured per Day (History):");
	for (int val : historyCuredPerDay)
	{
		printf(" %d", val);
	}
	printf("\n");
	return 0;
}


/*Authors:
- Alan Enrique Maldonado Navarro
- Guillermo Gonzalez Mena

Repository: https://github.com/DrN3MESiS/covid-19-spread-simulator
*/
